#include <iostream>
#include <sstream>
#include <iomanip>

const int width = 34;

static void cudaSafeCall(hipError_t err)
{
	if (err != hipSuccess)
	{
		std::cerr << hipGetErrorString(err) << std::endl;
		exit(1);
	}
}

static int getCoreNumPerSP(int major, int minor)
{
	switch (major)
	{
		case 2:
		{
			if (minor == 0)
			{
				return 32;
			}
			else if (minor == 1)
			{
				return 48;
			}
			else
			{
				return 0;
			}
		}
		case 3:
		{
			return 192;
		}
		case 5:
		{
			return 128;
		}
		case 6:
		{
			if (minor == 0)
			{
				return 64;
			}
			else if ((minor == 1) || (minor == 2))
			{
				return 128;
			}
			else
			{
				return 0;
			}
		}
		default:
		{
			return 0;
		}
	}
}

static std::string formatSize(size_t size)
{
	std::ostringstream stringStream;

	stringStream.precision(1);
	stringStream << std::fixed;

	if (size < 1024 * 10)
	{
		stringStream << size;
	}
	else if (size < 1024 * 1024 * 10)
	{
		stringStream << (float)size / 1024 << " KiB";
	}
	else if (size < size_t(1024) * size_t(1024) * size_t(1024) * size_t(10))
	{
		stringStream << (float)size / (1024 * 1024) << " MiB";
	}
	else
	{
		stringStream << (float)size / (1024 * 1024 * 1024) << " GiB";
	}
	return stringStream.str();
}

static void displayDeviceProperties(hipDeviceProp_t& prop, int device)
{
	std::cout << std::setw(width) << std::left << "GPU Device ID:" << device << std::endl;
	std::cout << std::setw(width) << std::left << "  Name:" << prop.name << std::endl;
	std::cout << std::setw(width) << std::left << "  Compute Capability:" << prop.major << "." << prop.minor << std::endl;
	std::cout << std::setw(width) << std::left << "  MultiProcessor(s):" << prop.multiProcessorCount << std::endl;
	std::cout << std::setw(width) << std::left << "  Cores Per MultiProcessor:" << getCoreNumPerSP(prop.major, prop.minor) << std::endl;
	std::cout << std::setw(width) << std::left << "  Max Threads Per MultiProcessor:" << prop.maxThreadsPerMultiProcessor << std::endl;
	std::cout << std::setw(width) << std::left << "  Clock Rate:" << prop.clockRate / (1000) << " MHz" << std::endl;
	std::cout << std::setw(width) << std::left << "  Warp Size:" << prop.warpSize << std::endl;
	std::cout << std::setw(width) << std::left << "  L2 Cache Size:" << formatSize(prop.l2CacheSize) << std::endl;
	std::cout << std::setw(width) << std::left << "  Global Memory Size:" << formatSize(prop.totalGlobalMem) << std::endl;
	std::cout << std::setw(width) << std::left << "  Constant Memory Size:" << formatSize(prop.totalConstMem) << std::endl;
	std::cout << std::setw(width) << std::left << "  One-Dimension Texture Size:" << prop.maxTexture1D << std::endl;
	std::cout << std::setw(width) << std::left << "  Two-Dimension Texture Size:" << prop.maxTexture2D[0] << " x "
			<< prop.maxTexture2D[1] << std::endl;
	std::cout << std::setw(width) << std::left << "  Three-Dimension Texture Size:" << prop.maxTexture3D[0] << " x "
			<< prop.maxTexture3D[1] << " x " << prop.maxTexture3D[2] << std::endl;
	std::cout << std::setw(width) << std::left << "  Shared Memory Size Per Block:" << formatSize(prop.sharedMemPerBlock) << std::endl;
	std::cout << std::setw(width) << std::left << "  Max Threads Per Block:" << prop.maxThreadsPerBlock << std::endl;
	std::cout << std::setw(width) << std::left << "  Registers Per Block:" << prop.regsPerBlock << std::endl;
	std::cout << std::setw(width) << std::left << "  Block Dimension:" << prop.maxThreadsDim[0] << " x "
			<< prop.maxThreadsDim[1] << " x " << prop.maxThreadsDim[2] << std::endl;
	std::cout << std::setw(width) << std::left << "  Grid Dimension:" << prop.maxGridSize[0] << " x "
			<< prop.maxGridSize[1] << " x " << prop.maxGridSize[2] << std::endl;

	return;
}

int main(void)
{
	int gpuCount = 0;
	cudaSafeCall(hipGetDeviceCount(&gpuCount));

	int runtimeVersion = 0;
	cudaSafeCall(hipRuntimeGetVersion(&runtimeVersion));

	int driverVersion = 0;
	cudaSafeCall(hipDriverGetVersion(&driverVersion));

	hipDeviceProp_t *prop = new hipDeviceProp_t[gpuCount];
	if (prop == NULL)
	{
		std::cout << "The memory is too small, and please enlarge it, thanks!" << std::endl;
		exit(1);
	}

	for (int i = 0; i < gpuCount; i++)
	{
		cudaSafeCall(hipGetDeviceProperties(prop + i, i));
	}

	std::cout << std::setw(width) << std::left << "CUDA Runtime Version:" << runtimeVersion / 1000 << "." << (runtimeVersion % 100) / 10 << std::endl;
	std::cout << std::setw(width) << std::left << "CUDA Driver Version:" << driverVersion / 1000 << "." << (driverVersion % 100) / 10 << std::endl;
	std::cout << std::setw(width) << std::left << "GPU(s):" << gpuCount << std::endl;

	for (int i = 0; i < gpuCount; i++)
	{
		displayDeviceProperties(prop[i], i);
	}

	delete[] prop;
	return 0;
}


