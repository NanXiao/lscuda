
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>

const int width = 25;

static void cudaSafeCall(hipError_t err)
{
	if (err != hipSuccess)
	{
		std::cerr << hipGetErrorString(err) << std::endl;
		exit(1);
	}
}

int main(void)
{
	int gpuCount = 0;

	cudaSafeCall(hipGetDeviceCount(&gpuCount));

	std::cout << std::setw(width) << std::left << "GPU(s):" << gpuCount << std::endl;
	return 0;
}


